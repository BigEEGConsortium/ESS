#include <time.h>
#include "nvmatrix.cuh"
#include "tsne.cuh"
#include "mex.h"


/* Implementation of t-SNE in CUDA (designed for Matlab).
 *
 *
 * (C) Laurens van der Maaten, 2010
 * University of California, San Diego
 *
 */
void tsne(float* inp_P, unsigned int N, float* mappedX, unsigned int no_dims) {

    /* Initialize some variables */
    int max_iter = 1000;
    float initial_momentum = 0.5f;
	float final_momentum = 0.8f;
    int momentum_switch_iter = 250;
    int lie_switch_iter = 100;
    float momentum = initial_momentum;    
	float eta = 500.0f;
	
    /* Fire up cublas */
    cutilSafeCall(hipSetDevice(cutGetMaxGflopsDeviceId()));
    hipblasStatus_t status = cublasInit();
    if(status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! cublas initialization failed\n");
    }
	NVMatrix::initDeviceProps();
    NVMatrix::initRandom(time(0));

    /* Print memory information */
    unsigned int freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    fprintf(stdout, "Running CUDA implementation of t-SNE...\n");
    fprintf(stdout, " - GPU memory is %d bytes (%d available).\n", totalMem, freeMem);
    fprintf(stdout, " - NOTE: This implementation does not show intermediate plots.\n");

    /* Copy data onto device, and make sure it is normalized */
    NVMatrix* Q = new NVMatrix(N, N);
    NVMatrix* P = new NVMatrix(true, inp_P, N, N);
	P->zeroDiagonal();
    Q->copyFromDevice(*P);
    P->add(Q->getTranspose());
    P->scale(1.0f / P->sum());
	P->addScalar(FLT_MIN);
	P->scale(4.0f);
	
    /* Initialize the solution */
    NVMatrix* Y = new NVMatrix(N, no_dims, false);
    Y->apply(NVMatrix::ZERO);
    Y->addGaussianNoise(0.0001f);

    /* Allocate some memory */
    NVMatrix* Qnum = new NVMatrix(N, N);
	NVMatrix* sum_Q = new NVMatrix(1, N);
	NVMatrix* sum_Y = new NVMatrix(1, N);
	NVMatrix* square_Y = new NVMatrix(N, no_dims);
	NVMatrix* dY = new NVMatrix(N, no_dims);
	NVMatrix* diffY = new NVMatrix(N, no_dims);
	NVMatrix* incY = new NVMatrix(N, no_dims);
    NVMatrix* gains = new NVMatrix(N, no_dims);
    NVMatrix* gains_update1 = new NVMatrix(N, no_dims);
	NVMatrix* gains_update2 = new NVMatrix(N, no_dims);
	incY->apply(NVMatrix::ZERO);
    gains->apply(NVMatrix::ONE);
	
    /* Perform updates */
    for(int iter = 0; iter < max_iter; iter++) {
		
		/* Create transposes, and stop early stopping */
		NVMatrix* Y_trans = &Y->getTranspose();
		NVMatrix* sum_Y_trans = &sum_Y->getTranspose();
		if(iter == lie_switch_iter) {
			P->scale(0.25f);
		}
        if(iter == momentum_switch_iter) {
            momentum = final_momentum;
        }
		
		/* Compute pairwise similarity matrix for the map */		
		square_Y->copyFromDevice(*Y);
		square_Y->apply(NVMatrix::SQUARE);
		square_Y->sum(1, *sum_Y);
		Y->rightMult(*Y_trans, -2.0f, *Qnum);
		Qnum->addVector(*sum_Y);
		Qnum->addVector(*sum_Y_trans);
		Qnum->apply(NVMatrix::STUDENT);
		Qnum->zeroDiagonal();
		Q->copyFromDevice(*Qnum);
		Q->scale(1.0f / Q->sum());
		
		/* Clean up memory */
		delete Y_trans;
		delete sum_Y_trans;
		
		/* Compute gradient */
		Q->add(*P, -1.0f, 1.0f);
		Q->eltWiseMult(*Qnum);     
		Q->sum(0, *sum_Q);
		Q->scale(-1.0f);
		Q->setDiagonal(*sum_Q);
		Q->rightMult(*Y, *dY);

        /* Update gains */
        dY->compareSigns(*incY, *gains_update1);
        gains_update1->addScalar(-1.0f); 
        gains_update1->scale(-1.0f);        
        dY->compareSigns(*incY, *gains_update2);
        gains_update2->eltWiseMult(*gains);
        gains_update2->scale(0.8f);        
        gains->addScalar(0.2f);
        gains->eltWiseMult(*gains_update1);
        gains->add(*gains_update2);
        
		/* Perform map update */
        dY->eltWiseMult(*gains);
		incY->add(*dY, momentum, -eta);
		Y->add(*incY);
		
		/* Print out progress */
		if((iter + 1) % 100 == 0) {
			Q->copyFromDevice(*Qnum);
			Q->scale(1.0f / Q->sum());
			Q->addScalar(FLT_MIN);
			P->eltWiseDivide(*Q, *Q);
			Q->apply(NVMatrix::LOG);
			Q->eltWiseMult(*P);
			Q->zeroDiagonal();
			float C = Q->sum();
			fprintf(stdout, "Iteration %d of %d: KL(P||Q) = %f\n", iter + 1, max_iter, C);
		}
	}
	
	/* Copy low-dimensional map to host */
	hipDeviceSynchronize();
	Y->getTranspose().copyToHost(mappedX, no_dims, N);
	
	/* Clean up some memory */
	delete Y;
	delete Q;
	delete Qnum;
	delete sum_Q;
	delete sum_Y;
	delete square_Y;
	delete dY;
	delete diffY;
	delete incY;
    delete gains;
    delete gains_update1;
    delete gains_update2;
	       
    /* Shut down cublas */
    NVMatrix::destroyRandom();
    status = cublasShutdown();
	if(status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! error while shutting down cublas\n");
	}
	hipDeviceReset();
}


/* Function call:
 *
 *		mappedX = tsne_p(P, labels, no_dims);
 *
 */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
 
    /* Check inputs */
    if(nrhs < 1) {
        mexErrMsgTxt("Function requires at least one input.");
    }
    if(!mxIsDouble(prhs[0])) {
        mexErrMsgTxt("Error in input (input data should be doubles).");
    }
	if(nrhs > 1 && !mxIsDouble(prhs[1])) {
		mexErrMsgTxt("Error in input (number of dimensions should be a double).");
	}
    
    /* Process inputs */
    double* inp_P = mxGetPr(prhs[0]);
    int no_dims   = nrhs > 2 ? (int) *mxGetPr(prhs[2]) : 2;
    if(mxGetN(prhs[2]) > 1 || mxGetM(prhs[2]) > 1) {
        mexErrMsgTxt("Specification of an initial solution is not supported by the CUDA implementation of t-SNE.");
    }
	int N = mxGetM(prhs[0]);
	if(mxGetN(prhs[0]) != N) {
		mexErrMsgTxt("Input similarities P should be a square matrix.");
	}
    
    /* Convert data to float */
    float* P = (float*) malloc(N * N * sizeof(float));
    for(int i = 0; i < N * N; i++) {
        P[i] = (float) inp_P[i];
    }
    
    /* Perform t-SNE */
    float* mappedX = (float*) malloc(N * no_dims * sizeof(float));
    tsne(P, N, mappedX, no_dims);

    /* Construct output matrix (transpose) */
    mwSize dims[2] = {N, no_dims};
    plhs[0] = mxCreateNumericArray(2, dims, mxSINGLE_CLASS, mxREAL);
    float* matlab_mappedX = (float*) mxGetPr(plhs[0]);
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < no_dims; j++) {
            matlab_mappedX[j * N + i] = mappedX[i * no_dims + j];
        }
    }

    /* Clean up memory */
    free(P);
    free(mappedX);
}
